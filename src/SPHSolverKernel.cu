#include "hip/hip_runtime.h"
#include "SPHSolverKernel.cuh"

int n_blocks;
int n_threads;

__device__ glm::ivec3 getHashPos_kernel(const glm::vec3& pos)
{
	return { pos.x / d_params.grid_cell, pos.y / d_params.grid_cell, pos.z / d_params.grid_cell };
}

__device__ uint getHashKey_kernel(const glm::ivec3& pos)
{
	return ((uint)(pos.x * 73856093) ^
			(uint)(pos.y * 19349663) ^
			(uint)(pos.z * 83492791)) % info::HASH_SIZE;
}

//__device__ void copyFromCuda_kernel(int n, glm::vec3* pos, glm::vec3* h_pos)
//{
//	return __device__ void();
//}

void computeBlocks(int n)
{
	n_threads = min(256, n);
	n_blocks = (n % n_threads != 0) ? (n / n_threads + 1) : (n / n_threads);

	cout << "Number of threads: " << n_threads << " Number of blocks: " << n_blocks << endl;
}

hipError_t setParams(info::SPHParams* params)
{
	hipError_t cuda_status;
	cuda_status = hipMalloc((void**)&d_params, sizeof(info::SPHParams));
	if (cuda_status != hipSuccess)
	{
		cout << "hipMalloc failed in setParams" << endl;
		return cuda_status;
	}

	cuda_status = hipMemcpyToSymbol((const void*)&d_params, params, sizeof(info::SPHParams));
	if (cuda_status != hipSuccess)
	{
		cout << "hipMemcpy failed in setParams: " << cuda_status << endl;
		return cuda_status;
	}
	
	return cuda_status;
}

hipError_t setHash(vector<int>& hash, vector<int>& neighbors)
{
	hipError_t cuda_status;
	cuda_status = hipMalloc((void**)&d_hash, sizeof(int) * hash.size());
	if (cuda_status != hipSuccess)
	{
		cout << "cudaMalloc1 failed in setHash" << endl;
		return cuda_status;
	}

	cuda_status = hipMalloc((void**)&d_neighbors, sizeof(int) * neighbors.size());
	if (cuda_status != hipSuccess)
	{
		cout << "cudaMalloc2 failed in setHash" << endl;
		return cuda_status;
	}

	cuda_status = hipMemcpy(d_hash, &hash[0], sizeof(int) * hash.size(), hipMemcpyHostToDevice);
	if (cuda_status != hipSuccess)
	{
		cout << "cudaMemcpy1 failed in setHash" << endl;
		return cuda_status;
	}

	cuda_status = hipMemcpy(d_neighbors, &neighbors[0], sizeof(int) * neighbors.size(), hipMemcpyHostToDevice);
	if (cuda_status != hipSuccess)
	{
		cout << "cudaMemcpy2 failed in setHash" << endl;
		return cuda_status;
	}

	return cuda_status;
}

hipError_t copyToCuda(
	int n,
	vector<glm::vec3>& pos, 
	vector<glm::vec3>& vel,
	vector<glm::vec3>& force,
	vector<float>& density,
	vector<float>& pressure)
{
	cout << "Copy to cuda : " << n << endl;

	hipError_t cuda_status;
	cuda_status = hipMalloc((void**)&d_pos, n * sizeof(glm::vec3));
	if (cuda_status != hipSuccess)
	{
		cout << "cudaMalloc1 failed in copyToCuda " << cuda_status << endl;
		return cuda_status;
	}

	cuda_status = hipMalloc((void**)&d_velocity, n * sizeof(glm::vec3));
	if (cuda_status != hipSuccess)
	{
		cout << "cudaMalloc2 failed in copyToCuda" << endl;
		return cuda_status;
	}

	cuda_status = hipMalloc((void**)&d_force, n * sizeof(glm::vec3));
	if (cuda_status != hipSuccess)
	{
		cout << "cudaMalloc3 failed in copyToCuda" << endl;
		return cuda_status;
	}

	cuda_status = hipMalloc((void**)&d_density, n * sizeof(float));
	if (cuda_status != hipSuccess)
	{
		cout << "cudaMalloc4 failed in copyToCuda" << endl;
		return cuda_status;
	}

	cuda_status = hipMalloc((void**)&d_pressure, n * sizeof(float));
	if (cuda_status != hipSuccess)
	{
		cout << "cudaMalloc5 failed in copyToCuda" << endl;
		return cuda_status;
	}

	cuda_status = hipMemcpy(d_pos, &pos[0], n * sizeof(glm::vec3), hipMemcpyHostToDevice);
	if (cuda_status != hipSuccess)
	{
		cout << "cudaMemcpy1 failed while copytocuda position" << endl;
		return cuda_status;
	}

	cuda_status = hipMemcpy(d_velocity, &vel[0], n * sizeof(glm::vec3), hipMemcpyHostToDevice);
	if (cuda_status != hipSuccess)
	{
		cout << "cudaMemcpy2 failed while copytocuda position" << endl;
		return cuda_status;
	}

	cuda_status = hipMemcpy(d_force, &force[0], n * sizeof(glm::vec3), hipMemcpyHostToDevice);
	if (cuda_status != hipSuccess)
	{
		cout << "cudaMemcpy3 failed while copytocuda position" << endl;
		return cuda_status;
	}

	cuda_status = hipMemcpy(d_density, &density[0], n * sizeof(float), hipMemcpyHostToDevice);
	if (cuda_status != hipSuccess)
	{
		cout << "cudaMemcpy4 failed while copytocuda density" << endl;
		return cuda_status;
	}

	cuda_status = hipMemcpy(d_pressure, &pressure[0], n * sizeof(float), hipMemcpyHostToDevice);
	if (cuda_status != hipSuccess)
	{
		cout << "cudaMemcpy5 failed while copytocuda pressure" << endl;
		return cuda_status;
	}

	return cuda_status;
}

hipError_t copyFromCuda(
	int n,
	vector<glm::vec3>& h_pos)
{
	hipError_t cuda_status;

	cuda_status = hipDeviceSynchronize();
	if (cuda_status != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after simulation!\n", cuda_status);
		assert(0);
	}

	//glm::vec3* h_pos;
	//cuda_status = hipMalloc((void**)&h_pos, n * sizeof(glm::vec3));
	//if (cuda_status != hipSuccess)
	//{
	//	cout << "cudaMalloc1 failed in copyFromCuda " << cuda_status << endl;
	//	assert(0);
	//}


	cuda_status = hipMemcpy(&h_pos[0], d_pos, n * sizeof(glm::vec3), hipMemcpyDeviceToHost);
	if (cuda_status != hipSuccess)
	{
		cout << "cudaMemcpy1 failed in copyFromCuda " << cuda_status << endl;
		return cuda_status;
	}

	//glm::vec3* host_pos = (glm::vec3*)malloc(pos.size() * sizeof(glm::vec3));
	//cout << "Pos size : " << d_pos[0].x << endl;
	//cuda_status = hipMemcpy(pos, d_pos, pos.size()*sizeof(glm::vec3), hipMemcpyDeviceToHost);

	hipFree(d_hash);
	hipFree(d_neighbors);

	return cuda_status;		
}

hipError_t simulateCuda(int n, float t, glm::vec3& box, vector<glm::vec3>& pos)
{
	//cout << "SimulateCuda " << n_blocks << " " << n_threads << endl;
	
	hipError_t cuda_status;

	fillHash_kernel << <n_blocks, n_threads >> > (n, d_hash, d_neighbors, d_pos);
	cuda_status = hipDeviceSynchronize();
	if (cuda_status != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after filling hash!\n", cuda_status);
		assert(0);
	}

	updateDensPress_kernel << <n_blocks, n_threads >> > (n, d_hash, d_neighbors, d_pos, d_density, d_pressure);
	cuda_status = hipDeviceSynchronize();
	if (cuda_status != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after update density and pressure!\n", cuda_status);
		assert( 0);
	}

	updateForce_kernel << <n_blocks, n_threads >> > (n, d_hash, d_neighbors, d_pos, d_velocity, d_force, d_density, d_pressure);
	cuda_status = hipDeviceSynchronize();
	if (cuda_status != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after udpate force!\n", cuda_status);
		assert(0);
	}

	updatePosition_kernel << <n_blocks, n_threads >> > (n, d_pos, d_velocity, d_force, d_density);
	cuda_status = hipDeviceSynchronize();
	if (cuda_status != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after udpate position!\n", cuda_status);
	}

	cuda_status = hipDeviceSynchronize();
	if (cuda_status != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after udpate!\n", cuda_status);
	}

	cuda_status = hipMemcpy(&pos[0], d_pos, n * sizeof(glm::vec3), hipMemcpyDeviceToHost);
	if (cuda_status != hipSuccess)
	{
		cout << "cudaMemcpy1 failed in copyFromCuda " << cuda_status << endl;
		return cuda_status;
	}

	hipFree(d_hash);
	hipFree(d_neighbors);

	return cuda_status;
}

hipError_t freeResources()
{
	//hipFree(&d_params);
	hipFree(d_hash);
	hipFree(d_neighbors);
	hipFree(d_pos);
	hipFree(d_velocity);
	hipFree(d_force);
	hipFree(d_density);
	hipFree(d_pressure);

	hipError_t cuda_status;
	cuda_status = hipDeviceSynchronize();
	if (cuda_status != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after update density and pressure!\n", cuda_status);
		assert(0);
	}
	
	return cuda_status;
}

__global__ void updateDensPress_kernel(
	int n,
	int* hash,
	int* neighbors,
	glm::vec3* pos,
	float* density,
	float* pressure)
{
	int id1 = blockIdx.x * blockDim.x + threadIdx.x;
	if (id1 >= n) return;

	int count = 0;
	float sum = 0.0f;
	glm::vec3 p1 = pos[id1];
	glm::ivec3 p1_grid = getHashPos_kernel(p1);
	for (int x = -1; x <= 1; x++)
	{
		for (int y = -1; y <= 1; y++)
		{
			for (int z = -1; z <= 1; z++)
			{
				glm::ivec3 near_pos = p1_grid + glm::ivec3(x, y, z);
				uint hash_key = getHashKey_kernel(near_pos);
				
				for (int i = hash[hash_key]; i < n * d_params.max_num_neighbors; i += n)
				{
					int id2 = neighbors[i];
					glm::vec3 p2 = pos[id2];

					const float r = glm::length(p2 - p1);
					const float r2 = r * r;
					//if (id1 == 10)
					//{
					//	printf("At %d sum : %f \n", n * d_params.max_num_neighbors, sum);
					//}
					if (r2 < d_params.H2 && id1 != id2)
					{						
						++count;
						float a = pow(d_params.H2 - r2, 3);
						sum += float(d_params.MASS * d_params.POLY6 * a);
						//if (id1 == 10)
						//{
						//	//printf("At %d count : %d, %f = %f * %f * %.10f \n",
						//	//		id1, count, 
						//	//		sum, d_params.MASS, d_params.POLY6, a);

						//	printf("{%.3f %.3f %.3f} - {%.3f %.3f %.3f} \n",
						//			p1.x, p1.y, p1.z, p2.x, p2.y, p2.z);
						//}
					}
				}
			}
		}
	}

	density[id1] = float(d_params.MASS * d_params.POLY6 * pow(d_params.H, 6)) + sum;
	pressure[id1] = d_params.K * (density[id1] - d_params.rDENSITY);

	//if (id1 == 10)
	//{
	//	printf("Final! At %d, Sum : %f density : %f pressure : %f \n", id1, sum, density[id1], pressure[id1]);
	//}
}

__global__ void updateForce_kernel(
	int n,
	int* hash,
	int* neighbors,
	glm::vec3* pos,
	glm::vec3* vel,
	glm::vec3* force,
	float* density,
	float* pressure)
{
	int id1 = blockIdx.x * blockDim.x + threadIdx.x;
	if (id1 >= n) return;

	float sum = 0.0f;
	glm::vec3 p1 = pos[id1];
	glm::ivec3 p1_grid = getHashPos_kernel(p1);
	force[id1] = glm::vec3(0.0f);

	for (int x = -1; x <= 1; x++)
	{
		for (int y = -1; y <= 1; y++)
		{
			for (int z = -1; z <= 1; z++)
			{
				glm::ivec3 near_pos = p1_grid + glm::ivec3(x, y, z);
				uint hash_key = getHashKey_kernel(near_pos);

				for (int i = hash[hash_key]; i < n * d_params.max_num_neighbors; i += n)
				{
					int id2 = neighbors[i];
					glm::vec3 p2 = pos[id2];

					const float r = glm::length(p2 - p1);
					const float r2 = r * r;
					if (r2 < d_params.H2 && id1 != id2)
					{
						glm::vec3 p_dir = glm::normalize(p2 - p1);

						float W = d_params.SPICKY * pow(d_params.H - r, 2);
						float temp = d_params.MASS * (pressure[id1] + pressure[id2]) / (2 * density[id1]);
						glm::vec3 a = -p_dir * temp;
						glm::vec3 f1 = a * W;

						glm::vec3 v_dir = vel[id2] - vel[id1];
						float W2 = d_params.SPICKY2 * (d_params.H - r);
						glm::vec3 b = d_params.VISC * d_params.MASS * (v_dir / density[id2]);
						glm::vec3 f2 = b * W2;

						force[id1] += f1 + f2;

						//if (id1 == 10)
						//{
						//	printf("At %d ", id1);
						//	
						//	printf("p_dir = (%f, %f, %f) = (%f, %f, %f) - (%f, %f, %f) \n", 
						//			p_dir.x, p_dir.y, p_dir.z, 
						//			p1.x, p1.y, p1.z,
						//			p2.x, p2.y, p2.z);

						//	printf("%f = %f * (%f - %f) / (2 * %f) \n",
						//			temp, d_params.MASS, pressure[id1], pressure[id2], density[id1]);

						//	printf("Force : (%f, %f, %f) = (%f, %f, %f) + (%f, %f, %f) \n", 
						//			force[id1].x, force[id1].y, force[id1].z, 
						//			f1.x, f1.y, f1.z, 
						//			f2.x, f2.y, f2.z);
						//}
					}
				}
			}
		}
	}

}

__global__ void updatePosition_kernel(
	int n,
	glm::vec3* pos,
	glm::vec3* vel,
	glm::vec3* force,
	float* density)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= n) return;
	
	vel[i] += d_params.t * (force[i] / density[i] + glm::vec3(0.0f, -9.8f, 0.0f));
	pos[i] += d_params.t * vel[i];
	
	//if (i == 10)
	//{
	//	printf("At %d ", i);
	//	
	//	printf("Pos : (%.3f, %.3f, %.3f)  Density : %.3f  Force : (%.3f, %.3f, %.3f) \n",
	//			pos[i].x, pos[i].y, pos[i].z,
	//			density[i],
	//			force[i].x, force[i].y, force[i].z);

	//	printf("Box : (%.3f, %.3f, %.3f) with %f \n", 
	//			d_params.box.x, d_params.box.y, d_params.box.z, d_params.grid_cell);

	//	printf("\n");
	//}
	//	printf("Box : {%.3f, %.3f, %.3f} with H : {%.3f}\n",
	//			d_params.box.x, d_params.box.y, d_params.box.z,
	//			d_params.H);
	//}
	
	if (pos[i].x > -d_params.H + d_params.box.x)
	{
		vel[i].x *= d_params.WALL;
		pos[i].x = -d_params.H + d_params.box.x;
	}
	if (pos[i].x < d_params.H - d_params.box.x)
	{
		vel[i].x *= d_params.WALL;
		pos[i].x = d_params.H - d_params.box.x;
	}

	if (pos[i].y > -d_params.H + d_params.box.y)
	{
		vel[i].y *= d_params.WALL;
		pos[i].y = -d_params.H + d_params.box.y;
	}
	if (pos[i].y < d_params.H)
	{
		vel[i].y *= d_params.WALL;
		pos[i].y = d_params.H;
	}

	if (pos[i].z > -d_params.H + d_params.box.z)
	{
		vel[i].z *= d_params.WALL;
		pos[i].z = -d_params.H + d_params.box.z;
	}
	if (pos[i].z < d_params.H - d_params.box.z)
	{
		vel[i].z *= d_params.WALL;
		pos[i].z = d_params.H - d_params.box.z;
	}
}

__global__ void fillHash_kernel(
	int n, 
	int* hash, 
	int* neighbors, 
	glm::vec3* pos)
{
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	if (id >= n) return;
	
	glm::vec3 p1 = pos[id];
	glm::ivec3 p1_grid = getHashPos_kernel(p1);
	uint hash_key = getHashKey_kernel(p1_grid);

	if (hash[hash_key] == -1)
	{
		hash[hash_key] = id;
		neighbors[id] = id;
	}
	else
	{
		int start = hash[hash_key];
		for (int i = start; i < n * d_params.max_num_neighbors; i += n)
		{
			const float r = glm::length(p1 - pos[start]);
			const float r2 = r * r;
			if (neighbors[i] == -1)
			{
				neighbors[i] = id;
				break;
			}
		}
	}

	//if (hash[hash_key] == 1000)
	//{
	//	printf("Pos : {%.2f, %.2f, %.2f} -> grid_pos : {%d, %d, %d} with hash_key : {%d}\n",
	//	p1.x, p1.y, p1.z, p1_grid.x, p1_grid.y, p1_grid.z, hash_key);
	//	
	//	printf("Hash : {%d}\n", hash[hash_key]);
	//}

}


//void fillHash(int n)
//{
//	fillHash_kernel<<<1, 1>>>(n);
//}
//
//__global__ void fillHash_kernel(int n)
//{
//	int i = blockIdx.x * blockDim.x + threadIdx.x;
//	if (i < n) return;
//
//	//FluidParticle* p = d_particles[i];
//
//	glm::vec3 pos = d_pos[i];
//	glm::ivec3 grid_pos = getHashPos(pos);
//	uint index = getHashKey(grid_pos);
//
//	if (d_hash[index] == nullptr)
//	{
//		d_hash[index] = new FluidParticle(pos);
//		d_hash[index]->m_next = nullptr;
//	}
//	else
//	{
//		FluidParticle* temp = d_hash[index];
//		d_hash[index] = new FluidParticle(pos);
//		d_hash[index]->m_next = temp;
//	}
//}
