#include "hip/hip_runtime.h"
#include "SPHSolverKernel.cuh"

int n_blocks;
int n_threads;

__device__ glm::ivec3 getHashPos_kernel(const glm::vec3& pos)
{
	return { pos.x / d_params.grid_cell, pos.y / d_params.grid_cell, pos.z / d_params.grid_cell };
}

__device__ uint getHashKey_kernel(const glm::ivec3& pos)
{
	return ((uint)(pos.x * 73856093) ^
			(uint)(pos.y * 19349663) ^
			(uint)(pos.z * 83492791)) % info::HASH_SIZE;
}

void computeBlocks(int n)
{
	n_threads = min(256, n);
	n_blocks = (n % n_threads != 0) ? (n / n_threads + 1) : (n / n_threads);

	cout << "Number of threads: " << n_threads << " Number of blocks: " << n_blocks << endl;
}

hipError_t setParams(info::SPHParams* params)
{
	hipError_t cuda_status;
	cuda_status = hipMalloc((void**)&d_params, sizeof(info::SPHParams));
	if (cuda_status != hipSuccess)
	{
		cout << "hipMalloc failed in setParams" << endl;
		return cuda_status;
	}

	cuda_status = hipMemcpyToSymbol((const void*)&d_params, params, sizeof(info::SPHParams));
	if (cuda_status != hipSuccess)
	{
		cout << "hipMemcpy failed in setParams: " << cuda_status << endl;
		return cuda_status;
	}
	
	//cout << d_params.H << endl;

	return cuda_status;
}

hipError_t setHash(vector<int>& hash, vector<int>& neighbors)
{
	hipError_t cuda_status;
	cuda_status = hipMalloc((void**)&d_hash, sizeof(int) * hash.size());
	if (cuda_status != hipSuccess)
	{
		cout << "cudaMalloc1 failed in setHash" << endl;
		return cuda_status;
	}

	cuda_status = hipMalloc((void**)&d_neighbors, sizeof(int) * neighbors.size());
	if (cuda_status != hipSuccess)
	{
		cout << "cudaMalloc2 failed in setHash" << endl;
		return cuda_status;
	}

	cuda_status = hipMemcpy(d_hash, &hash[0], sizeof(int) * hash.size(), hipMemcpyHostToDevice);
	if (cuda_status != hipSuccess)
	{
		cout << "cudaMemcpy1 failed in setHash" << endl;
		return cuda_status;
	}

	cuda_status = hipMemcpy(d_neighbors, &neighbors[0], sizeof(int) * neighbors.size(), hipMemcpyHostToDevice);
	if (cuda_status != hipSuccess)
	{
		cout << "cudaMemcpy2 failed in setHash" << endl;
		return cuda_status;
	}

	return cuda_status;
}

hipError_t copyToCuda(
	int n,
	vector<glm::vec3>& pos, 
	vector<glm::vec3>& vel,
	vector<glm::vec3>& force,
	vector<float>& density,
	vector<float>& pressure)
{
	cout << "Copy to cuda : " << n << endl;

	hipError_t cuda_status;
	cuda_status = hipMalloc((void**)&d_pos, n * sizeof(glm::vec3));
	if (cuda_status != hipSuccess)
	{
		cout << "cudaMalloc1 failed in copyToCuda " << cuda_status << endl;
		return cuda_status;
	}

	cuda_status = hipMalloc((void**)&d_velocity, n * sizeof(glm::vec3));
	if (cuda_status != hipSuccess)
	{
		cout << "cudaMalloc2 failed in copyToCuda" << endl;
		return cuda_status;
	}

	cuda_status = hipMalloc((void**)&d_force, n * sizeof(glm::vec3));
	if (cuda_status != hipSuccess)
	{
		cout << "cudaMalloc3 failed in copyToCuda" << endl;
		return cuda_status;
	}

	cuda_status = hipMalloc((void**)&d_density, n * sizeof(float));
	if (cuda_status != hipSuccess)
	{
		cout << "cudaMalloc4 failed in copyToCuda" << endl;
		return cuda_status;
	}

	cuda_status = hipMalloc((void**)&d_pressure, n * sizeof(float));
	if (cuda_status != hipSuccess)
	{
		cout << "cudaMalloc5 failed in copyToCuda" << endl;
		return cuda_status;
	}

	cuda_status = hipMemcpy(d_pos, &pos[0], n * sizeof(glm::vec3), hipMemcpyHostToDevice);
	if (cuda_status != hipSuccess)
	{
		cout << "cudaMemcpy1 failed while copytocuda position" << endl;
		return cuda_status;
	}

	cuda_status = hipMemcpy(d_velocity, &vel[0], n * sizeof(glm::vec3), hipMemcpyHostToDevice);
	if (cuda_status != hipSuccess)
	{
		cout << "cudaMemcpy2 failed while copytocuda position" << endl;
		return cuda_status;
	}

	cuda_status = hipMemcpy(d_force, &force[0], n * sizeof(glm::vec3), hipMemcpyHostToDevice);
	if (cuda_status != hipSuccess)
	{
		cout << "cudaMemcpy3 failed while copytocuda position" << endl;
		return cuda_status;
	}

	cuda_status = hipMemcpy(d_density, &density[0], n * sizeof(float), hipMemcpyHostToDevice);
	if (cuda_status != hipSuccess)
	{
		cout << "cudaMemcpy4 failed while copytocuda density" << endl;
		return cuda_status;
	}

	cuda_status = hipMemcpy(d_pressure, &pressure[0], n * sizeof(float), hipMemcpyHostToDevice);
	if (cuda_status != hipSuccess)
	{
		cout << "cudaMemcpy5 failed while copytocuda pressure" << endl;
		return cuda_status;
	}

	return cuda_status;
}

hipError_t copyFromCuda(
	int n,
	vector<glm::vec3>& h_pos)
{
	hipError_t cuda_status;

	cuda_status = hipDeviceSynchronize();
	if (cuda_status != hipSuccess) {
		cout << "hipDeviceSynchronize returned error code after simulation!: " << cuda_status << endl;
		return cuda_status;
	}

	cuda_status = hipMemcpy(&h_pos[0], d_pos, n * sizeof(glm::vec3), hipMemcpyDeviceToHost);
	if (cuda_status != hipSuccess)
	{
		cout << "cudaMemcpy1 failed in copyFromCuda " << cuda_status << endl;
		return cuda_status;
	}

	hipFree(d_hash);
	hipFree(d_neighbors);

	return cuda_status;		
}

hipError_t simulateCuda(int n, float t, vector<glm::vec3>& pos)
{
	hipError_t cuda_status;

	fillHash_kernel << <n_blocks, n_threads >> > (n, d_hash, d_neighbors, d_pos);
	cuda_status = hipDeviceSynchronize();
	if (cuda_status != hipSuccess) {
		cout << "hipDeviceSynchronize returned error code after filling hash!: " << cuda_status << endl;
		return cuda_status;
	}

	updateDensPress_kernel << <n_blocks, n_threads >> > (n, d_hash, d_neighbors, d_pos, d_density, d_pressure);
	cuda_status = hipDeviceSynchronize();
	if (cuda_status != hipSuccess) {
		cout << "hipDeviceSynchronize returned error code after update density and pressure!: " << cuda_status << endl;
		return cuda_status;
	}

	updateForce_kernel << <n_blocks, n_threads >> > (n, d_hash, d_neighbors, d_pos, d_velocity, d_force, d_density, d_pressure);
	cuda_status = hipDeviceSynchronize();
	if (cuda_status != hipSuccess) {
		cout << "hipDeviceSynchronize returned error code after udpate force!: " << cuda_status << endl;
		return cuda_status;
	}

	updatePosition_kernel << <n_blocks, n_threads >> > (n, d_pos, d_velocity, d_force, d_density);
	cuda_status = hipDeviceSynchronize();
	if (cuda_status != hipSuccess) {
		cout << "hipDeviceSynchronize returned error code after udpate position!: " << cuda_status << endl;
		return cuda_status;

	}

	cuda_status = hipMemcpy(&pos[0], d_pos, n * sizeof(glm::vec3), hipMemcpyDeviceToHost);
	if (cuda_status != hipSuccess)
	{
		cout << "cudaMemcpy1 failed in copyFromCuda " << cuda_status << endl;
		return cuda_status;
	}

	hipFree(d_hash);
	hipFree(d_neighbors);

	return cuda_status;
}

hipError_t freeResources()
{
	//hipFree(&d_params);
	hipFree(d_hash);
	hipFree(d_neighbors);
	hipFree(d_pos);
	hipFree(d_velocity);
	hipFree(d_force);
	hipFree(d_density);
	hipFree(d_pressure);

	hipError_t cuda_status;
	cuda_status = hipDeviceSynchronize();
	if (cuda_status != hipSuccess) {
		cout << "hipDeviceSynchronize returned error code after free resources!: " << cuda_status << endl;
		return cuda_status;
	}
	
	return cuda_status;
}

__global__ void updateDensPress_kernel(
	int n,
	int* hash,
	int* neighbors,
	glm::vec3* pos,
	float* density,
	float* pressure)
{
	int id1 = blockIdx.x * blockDim.x + threadIdx.x;
	if (id1 >= n) return;

	int count = 0;
	float sum = 0.0f;
	glm::vec3 p1 = pos[id1];
	glm::ivec3 p1_grid = getHashPos_kernel(p1);
	for (int x = -1; x <= 1; x++)
	{
		for (int y = -1; y <= 1; y++)
		{
			for (int z = -1; z <= 1; z++)
			{
				glm::ivec3 near_pos = p1_grid + glm::ivec3(x, y, z);
				uint hash_key = getHashKey_kernel(near_pos);
				
				for (int i = hash[hash_key]; i < n * d_params.max_num_neighbors; i += n)
				{
					int id2 = neighbors[i];
					glm::vec3 p2 = pos[id2];

					const float r = glm::length(p2 - p1);
					const float r2 = r * r;
					if (r2 < d_params.H2 && id1 != id2)
					{						
						++count;
						float a = pow(d_params.H2 - r2, 3);
						sum += float(d_params.MASS * d_params.POLY6 * a);
					}
				}
			}
		}
	}

	density[id1] = float(d_params.MASS * d_params.POLY6 * pow(d_params.H, 6)) + sum;
	pressure[id1] = d_params.K * (density[id1] - d_params.rDENSITY);
}

__global__ void updateForce_kernel(
	int n,
	int* hash,
	int* neighbors,
	glm::vec3* pos,
	glm::vec3* vel,
	glm::vec3* force,
	float* density,
	float* pressure)
{
	int id1 = blockIdx.x * blockDim.x + threadIdx.x;
	if (id1 >= n) return;

	float sum = 0.0f;
	glm::vec3 p1 = pos[id1];
	glm::ivec3 p1_grid = getHashPos_kernel(p1);
	force[id1] = glm::vec3(0.0f);

	for (int x = -1; x <= 1; x++)
	{
		for (int y = -1; y <= 1; y++)
		{
			for (int z = -1; z <= 1; z++)
			{
				glm::ivec3 near_pos = p1_grid + glm::ivec3(x, y, z);
				uint hash_key = getHashKey_kernel(near_pos);

				for (int i = hash[hash_key]; i < n * d_params.max_num_neighbors; i += n)
				{
					int id2 = neighbors[i];
					glm::vec3 p2 = pos[id2];

					const float r = glm::length(p2 - p1);
					const float r2 = r * r;
					if (r2 < d_params.H2 && id1 != id2)
					{
						glm::vec3 p_dir = glm::normalize(p2 - p1);

						float W = d_params.SPICKY * pow(d_params.H - r, 2);
						float temp = d_params.MASS * (pressure[id1] + pressure[id2]) / (2 * density[id1]);
						glm::vec3 a = -p_dir * temp;
						glm::vec3 f1 = a * W;

						glm::vec3 v_dir = vel[id2] - vel[id1];
						float W2 = d_params.SPICKY2 * (d_params.H - r);
						glm::vec3 b = d_params.VISC * d_params.MASS * (v_dir / density[id2]);
						glm::vec3 f2 = b * W2;

						force[id1] += f1 + f2;
					}
				}
			}
		}
	}

}

__global__ void updatePosition_kernel(
	int n,
	glm::vec3* pos,
	glm::vec3* vel,
	glm::vec3* force,
	float* density)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= n) return;
	
	vel[i] += d_params.t * (force[i] / density[i] + glm::vec3(0.0f, -9.8f, 0.0f));
	pos[i] += d_params.t * vel[i];
	
	if (pos[i].x > -d_params.H + d_params.max_box.x)
	{
		vel[i].x *= d_params.WALL;
		pos[i].x = -d_params.H + d_params.max_box.x;
	}
	if (pos[i].x < d_params.H + d_params.min_box.x)
	{
		vel[i].x *= d_params.WALL;
		pos[i].x = d_params.H + d_params.min_box.x;
	}

	if (pos[i].y > -d_params.H + d_params.max_box.y)
	{
		vel[i].y *= d_params.WALL;
		pos[i].y = -d_params.H + d_params.max_box.y;
	}
	if (pos[i].y < d_params.H + d_params.min_box.y)
	{
		vel[i].y *= d_params.WALL;
		pos[i].y = d_params.H + d_params.min_box.y;
	}

	if (pos[i].z > -d_params.H + d_params.max_box.z)
	{
		vel[i].z *= d_params.WALL;
		pos[i].z = -d_params.H + d_params.max_box.z;
	}
	if (pos[i].z < d_params.H + d_params.min_box.z)
	{
		vel[i].z *= d_params.WALL;
		pos[i].z = d_params.H + d_params.min_box.z;
	}
}

__global__ void fillHash_kernel(
	int n, 
	int* hash, 
	int* neighbors, 
	glm::vec3* pos)
{
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	if (id >= n) return;
	
	glm::vec3 p1 = pos[id];
	glm::ivec3 p1_grid = getHashPos_kernel(p1);
	uint hash_key = getHashKey_kernel(p1_grid);

	if (hash[hash_key] == -1)
	{
		hash[hash_key] = id;
		neighbors[id] = id;
	}
	else
	{
		int start = hash[hash_key];
		for (int i = start; i < n * d_params.max_num_neighbors; i += n)
		{
			const float r = glm::length(p1 - pos[start]);
			const float r2 = r * r;
			if (neighbors[i] == -1)
			{
				neighbors[i] = id;
				break;
			}
		}
	}
}